#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/sequence.h>
#include <thrust/generate.h>
#include <thrust/detail/type_traits.h>

#include <hip/hip_runtime_api.h>

#include <algorithm>
#include <time.h>
#include <limits.h>

//#define RADIX 4294967296
#define RADIX 2147483658
//#define numElements 1048576
#define numElements 2048
#define numIterations 10

#define BLOCKSIZE 32
#define BINNUM 16
void 
sequentialSort(int *unsorted, int *sorted)
{
   int *count, *prefix;

  // count number of entries for each value
  count = (int *) malloc (RADIX*sizeof(int));
  for (unsigned int i=0; i<RADIX; i++) count[i]=0;
  for (int i=0; i<numElements; i++) {
    count[unsorted[i]]++;
  }

  // prefix sum of count
  prefix = (int *) malloc (RADIX*sizeof(int));
  prefix[0] = 0;
  for (unsigned int i=1; i<RADIX; i++) {
    prefix[i] = prefix[i-1] + count[i];
  }
  
  // generate result

  int curr = 0;
  for (unsigned int i=0; i<RADIX; i++) {
    for (int j=0; j<count[i]; j++) {
      sorted[curr++] = i;
    }
  }

}

// for 2 bins
__device__ int getBit(unsigned int &num, int pos)
{
  return (num >> (pos)) & (0x1);
}

// for 16 bins
__device__ int getBinIndex(unsigned int &num, int pos)
{ 
  return (num >> (pos)) & (0xf);
}

__device__ void reductionSum(int *s_data, int *sum, int tx)
{
 // printf("jc\n");
  //int tx = threadIdx.x;

  //__shared__ int tile[2][BLOCKSIZE];

  //tile[0][tx] = s_data[tx];
  //tile[1][tx] = s_data[BLOCKSIZE+tx];

//  for(int i=0;i<BLOCKSIZE;i++)
 // {
 //   printf("%d\n",s_data[i]);
 // }

  // blocksize must be power of 2
  for(int i = BLOCKSIZE; i > 0; i = i >> 1)
  {
    if(tx < i)
    {
      s_data[tx] += s_data[i+tx];
     // s_data[tx+BLOCKSIZE] += s_data[BLOCKSIZE+i+tx];
    }
      __syncthreads();
  }

  if(tx == 0)
  {
    sum[0] = s_data[0];
   // sum[1] = s_data[BLOCKSIZE];
  }
  

}
__global__ void global_radixsort(unsigned int *d_keys,
    int pos, int *blockCount, int numBlocks)
{
  int tx = threadIdx.x;
  int idx = blockIdx.x * blockDim.x + tx;

  // may be no need to use shared memory
 // __shared__ unsigned int tile[BLOCKSIZE];
 // tile[tx] = d_keys[i];

  __shared__ int perBlockCount[BINNUM*BLOCKSIZE];

  for(int i=0;i<BINNUM;++i)
    perBlockCount[i*BLOCKSIZE+tx] = 0;
  
  __syncthreads();
  
  int binIdx = getBinIndex(d_keys[idx], pos);

  perBlockCount[binIdx*BLOCKSIZE+tx] = 1;
  //printf("%d,%d\n",perBlockCount[zoo*BLOCKSIZE+tx],perBlockCount[(1-zoo)*BLOCKSIZE+tx]);
  __syncthreads();

  __shared__ int bin[BINNUM][BLOCKSIZE];
    
  for(int i=0;i<BINNUM;++i)
  {
    bin[i][tx] = perBlockCount[tx+i*BLOCKSIZE];
  }
  __syncthreads();
  
  for(int i = BLOCKSIZE/2; i > 0; i >>= 1)
  {
    if(tx < i)
    {
      for(int i=0;i<BINNUM;++i)
      {
        bin[i][tx] = bin[i][tx]+ bin[i][i+tx];
      }
    }
     __syncthreads();
  }

  if(tx == 0)
  {
    for(int i=0;i<BINNUM;++i)
    {
      blockCount[blockIdx.x+i*numBlocks] = bin[i][0];
    }
    //printf("Block %d, bin0--%d, bin1--%d\n",blockIdx.x,
    //  blockCount[blockIdx.x],blockCount[blockIdx.x+numBlocks]);
  }
}

__global__ void combineBlockCount(int *s_data, int *g_data)
{

  int tx = threadIdx.x;
  int idx = blockIdx.x * blockDim.x + tx;
  
  extern __shared__ int data[];
  data[tx] = s_data[idx];
  __syncthreads();

  for(int i=blockDim.x/2; i > 0; i >>= 1)
  {
    if(tx < i)
      data[tx] += data[i+tx];

    __syncthreads();
  }

  if(tx == 0)
  {
    g_data[blockIdx.x] = data[0];
  }
}

__global__ void prefixSumBottomUp(int *s_data, int *g_data)
{
  int tx = threadIdx.x;
  int idx = blockIdx.x*blockDim.x+tx;

  extern  __shared__ int tile[];
  tile[tx] = s_data[idx];
  //tile[tx+1] = s_data[idx+1];

  int data_size = blockDim.x;

  //tile[2*tx+1] += tile[2*tx];
  // bottom-up phase
  for(int i=2, size=data_size/2; i <= data_size; i <<= 1, size>>=1)
  {
    if( tx < size)
      tile[(i*tx+i-1)] += tile[i*tx+i>>1-1];
    __syncthreads();
  }
  tile[data_size-1] = 0;

  __syncthreads();

  g_data[tx] = tile[tx];
 // printf("tx:%d, %d\n", tx, g_data[tx]);

}
__global__ void prefixSumTopDown(int *s_data, int *g_data)
{
  int tx = threadIdx.x;
  int idx = blockIdx.x*blockDim.x+tx;

  extern  __shared__ int tile[]; 
  tile[tx] = s_data[idx];

  int data_size = blockDim.x;
  
  // top-down phase
  for(int i=data_size,size=1; i>=2; i>>=1,size<<=1)
  {
    if(tx<size)
    {
      int tmp = tile[i*tx+i-1];
      //printf("i>>1:%d, i*tx:%d, i*tx+i>>1-1:%d\n", i>>1, i*tx, 0+1-1);
      tile[i*tx+i-1] += tile[i*tx+(i>>1)-1];
      tile[i*tx+(i>>1)-1] = tmp;
     // printf("tile[%d]:%d,tile[%d]:%d\n", i*tx+i-1, tile[i*tx+i-1], i*tx+i>>1-1,
     //     tile[i*tx+i>>1-1]);
    }
     __syncthreads();
  }

  g_data[tx] = tile[tx];
  //printf("tx:%d, %d\n", tx, g_data[tx]);
}

__global__ void scatter(unsigned int *s_data, unsigned int *g_data, int *prefixSum,
    int pos)
{
  int tx = threadIdx.x;
  //int idx = blockIdx.x*blockDim.x + tx;

  int idx = prefixSum[tx];
  //extern  __shared__ int count[]; // size = BINNUM

  //if(tx < binsize)
  //{
  //  count[tx] = prefixSum[tx];
  //  //printf("tx:%d,%d\n", tx, count[tx]);
  //}

  //__syncthreads();

  for (int i=0;i<numElements;++i)
  {

    unsigned int key = s_data[i];
    int bit = getBit(key, pos);
    if(tx == bit)
    {
      g_data[idx] = s_data[i];
      idx++;
    }
  }

  //printf("out idx:%d\n", out_idx);
  //count[out_idx]++; //atomic
}
__host__ void host_radixsort(unsigned int *h_keys, unsigned int *h_sorted)
{
  unsigned int *d_keys;
  //unsigned int *d_keysSorted;
  unsigned int numbytes = numElements*sizeof(unsigned int);

  checkCudaErrors(hipMalloc((void **) &d_keys, numbytes));
  checkCudaErrors(hipMemcpy(d_keys, h_keys, numbytes, hipMemcpyHostToDevice));
 // checkCudaErrors(hipMalloc((void**)&d_keysSorted, numbytes));
 // checkCudaErrors(hipMemset(d_keysSorted, 0, numbytes));

  //int perBlockCount[2*BLOCKSIZE];
  int numBlocks = numElements / BLOCKSIZE;
 
  hipEvent_t my_start_event, my_stop_event;
  checkCudaErrors(hipEventCreate(&my_start_event));
  checkCudaErrors(hipEventCreate(&my_stop_event));
 
  checkCudaErrors(hipEventRecord(my_start_event, 0));
  int *blockCount;
  checkCudaErrors(hipMalloc((void**)&blockCount, 2*numBlocks*sizeof(int)));
  int *overallCount;
  checkCudaErrors(hipMalloc((void**)&overallCount, BINNUM*sizeof(int)));
  int *bottomUpResult;
  checkCudaErrors(hipMalloc((void**)&bottomUpResult, BINNUM*sizeof(int)));
  int *prefixSumArray;
  checkCudaErrors(hipMalloc((void**)&prefixSumArray, BINNUM*sizeof(int)));

  unsigned int *keys_tmp;
  checkCudaErrors(hipMalloc((void**)&keys_tmp, numElements*sizeof(int)));

  for(int i = 0; i < 32; i+=4)
  {
    global_radixsort<<<numBlocks, BLOCKSIZE>>>(d_keys, i, blockCount, numBlocks);
    // combine per block counting
    //printf("jc\n");
    combineBlockCount<<<BINNUM, numBlocks, numBlocks*sizeof(int)>>>(blockCount, overallCount);
    // prefix sum
    
    prefixSumBottomUp<<<1, BINNUM, BINNUM*sizeof(int)>>>(overallCount,
        bottomUpResult);

    prefixSumTopDown<<<1, BINNUM, BINNUM*sizeof(int)>>>(bottomUpResult,
        prefixSumArray);
    //scatter to d_keys
    scatter<<<1, BINNUM>>>(d_keys, keys_tmp, prefixSumArray,
        i);
    checkCudaErrors(hipMemcpy(d_keys, keys_tmp, numElements*sizeof(int),
          hipMemcpyDeviceToDevice));

  }

  checkCudaErrors(hipEventRecord(my_stop_event, 0));
  checkCudaErrors(hipEventSynchronize(my_stop_event));
  float my_time = 0;
  checkCudaErrors(hipEventElapsedTime(&my_time, my_start_event, my_stop_event));
  my_time /= 1.0e3f;
  printf("radixSort (MyTest), Throughput = %.4f KElements/s, Time = %.5f s, Size = %u elements\n",
           1.0e-3f * numElements / my_time, my_time, numElements);

#define testreorder

#ifdef testblockcount
  int *hostBlockCount;
  hostBlockCount = (int*)malloc(2*numBlocks*sizeof(int));
  checkCudaErrors(hipMemcpy(hostBlockCount, blockCount,
        2*numBlocks*sizeof(int), hipMemcpyDeviceToHost));
  for(int i = 0;i<numBlocks;++i)
    printf("block %d: bin0:%d, bin1:%d\n",i, hostBlockCount[i],
      hostBlockCount[i+numBlocks]);
#endif
  
#ifdef testoverallcount
  int *hostOverallCount;
  hostOverallCount = (int*)malloc(BINNUM*sizeof(int));
  checkCudaErrors(hipMemcpy(hostOverallCount, overallCount,
        BINNUM*sizeof(int), hipMemcpyDeviceToHost));

  for(int i=0; i < BINNUM;++i)
    printf("bin%d, %d\n", i, hostOverallCount[i]);
#endif

#ifdef testprefixsum
  int *hostPrefixSum;
  hostPrefixSum = (int*)malloc(BINNUM*sizeof(int));
  checkCudaErrors(hipMemcpy(hostPrefixSum, prefixSumArray,
        BINNUM*sizeof(int), hipMemcpyDeviceToHost));
  
  for(int i=0; i < BINNUM;++i)
    printf("prefix%d, %d\n", i, hostPrefixSum[i]);
#endif

#ifdef testreorder
  unsigned int *hostKeys;
  hostKeys = (unsigned int*)malloc(numElements*sizeof(int));
  checkCudaErrors(hipMemcpy(hostKeys, d_keys,
        numElements*sizeof(int), hipMemcpyDeviceToHost));
  
  for(int i=0; i < numElements;++i)
    printf("%d\n",hostKeys[i]);
#endif



  checkCudaErrors(hipDeviceSynchronize());
  //checkCudaErrors(hipMemcpy(d_keysSorted, h_sorted, numbytes, hipMemcpyDeviceToHost));

  checkCudaErrors(hipFree(d_keys));
  //checkCudaErrors(hipFree(d_keysSorted));
  



}

int
main(int argc, char **argv)
{
  int *unsorted, *sorted;

  // initialize list.  Value in range 0..RADIX
  unsorted = (int *) malloc (numElements*sizeof(int));
  sorted = (int *) malloc (numElements*sizeof(int));
  for (int i=0; i<numElements; i++) {
    unsorted[i] = (int) (rand() % RADIX);
    //unsorted[i] = 1;
  }

  //initialize list for Thrust
  thrust::host_vector<int> h_keys(numElements);
  thrust::host_vector<int> h_keysSorted(numElements);
  for (int i = 0; i < (int)numElements; i++)
     h_keys[i] = unsorted[i];

  // SEQUENTIAL RUN
  hipEvent_t seq_start_event, seq_stop_event;
  checkCudaErrors(hipEventCreate(&seq_start_event));
  checkCudaErrors(hipEventCreate(&seq_stop_event));
  checkCudaErrors(hipEventRecord(seq_start_event, 0));

  // TODO: THIS TAKES A FEW MINUTES AND SHOULD BE COMMENTED OUT FOR TESTING
  //(voir) sequentialSort(unsorted,sorted);

  //checkCudaErrors(hipEventRecord(seq_stop_event, 0));
  //checkCudaErrors(hipEventSynchronize(seq_stop_event));

  //float seq_time = 0;
  //checkCudaErrors(hipEventElapsedTime(&seq_time, seq_start_event, seq_stop_event));
  //seq_time /= 1.0e3f;
  //printf("radixSort (SEQ), Throughput = %.4f KElements/s, Time = %.5f s, Size = %u elements\n",
  //         1.0e-3f * numElements / seq_time, seq_time, numElements);


  // THRUST IMPLEMENTATION
  // copy onto GPU
  //thrust::device_vector<int> d_keys;
  //  
  //hipEvent_t start_event, stop_event;
  //checkCudaErrors(hipEventCreate(&start_event));
  //checkCudaErrors(hipEventCreate(&stop_event));

  //float totalTime = 0;
  //// run multiple iterations to compute an average sort time
  //for (int i = 0; i < numIterations; i++) {
  //      // reset data before sort
  //      d_keys= h_keys;

  //      checkCudaErrors(hipEventRecord(start_event, 0));

  //      thrust::sort(d_keys.begin(), d_keys.end());

  //      checkCudaErrors(hipEventRecord(stop_event, 0));
  //      checkCudaErrors(hipEventSynchronize(stop_event));

  //      float time = 0;
  //      checkCudaErrors(hipEventElapsedTime(&time, start_event, stop_event));
  //      totalTime += time;
  //  }

  //  totalTime /= (1.0e3f * numIterations);
  //  printf("radixSort in THRUST, Throughput = %.4f MElements/s, Time = %.5f s, Size = %u elements\n",
  //         1.0e-6f * numElements / totalTime, totalTime, numElements);

  //  getLastCudaError("after radixsort");

  //  // Get results back to host for correctness checking
  //  thrust::copy(d_keys.begin(), d_keys.end(), h_keysSorted.begin());

  //  getLastCudaError("copying results to host memory");

  //  // Check results
  //  bool bTestResult = thrust::is_sorted(h_keysSorted.begin(), h_keysSorted.end());

  //  checkCudaErrors(hipEventDestroy(start_event));
  //  checkCudaErrors(hipEventDestroy(stop_event));

  //  if (bTestResult) printf("THRUST: VALID!\n");

    // COMPARE SEQUENTIAL WITH THRUST
   //bTestResult = true;
   //for (int i = 0; i < (int)numElements; i++) {
   //  if (h_keysSorted[i] != sorted[i]) {
   //    bTestResult = false;
   //    break;
   //  }
   //}
   //if (bTestResult) printf("SEQ: VALID!\n");

   // TODO: NOW ADD YOUR OWN CODE, TIME AND VALIDATE AGAINST SEQUENTIAL


  //unsigned int *my_sorted;
  //my_sorted = (unsigned int *) malloc (numElements*sizeof(unsigned int));
  host_radixsort((unsigned int*)unsorted,(unsigned int*)sorted);

}

